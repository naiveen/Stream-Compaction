#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#define blockSize 512
namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        __global__ void kernelPreScan(int N,int d, int* dev_odata, int* dev_idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= N ) {
                return;
            }
            if (index >= pow(2, d-1)) {
                dev_odata[index] = dev_idata[index] + dev_idata[index - int(pow(2, d - 1))];
                
            }
            else {
                dev_odata[index] = dev_idata[index];
            }
            
        }
        void scan(int n, int *odata, const int *idata) {
            int* dev_idata;
            int* dev_odata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            timer().startGpuTimer();
            // TODO
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            for (auto d = 1; d <= ilog2ceil(n); d++) {
                kernelPreScan <<<fullBlocksPerGrid, blockSize >> > (n, d, dev_odata, dev_idata);
                hipMemcpy(dev_idata, dev_odata, n * sizeof(int), hipMemcpyDeviceToDevice);
            }
            timer().endGpuTimer();
            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
        }
    }
}
