#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            thrust::device_vector<int> dv_in(idata, idata+n);
            dv_in.push_back(0);
            thrust::device_vector<int> dv_out(n+1);
            
            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            //thrust::exclusive_scan(dv_in, dv_in +n, dv_out);
            timer().endGpuTimer();
            thrust::copy(dv_out.begin()+1, dv_out.end(), odata);
        }
    }
}
